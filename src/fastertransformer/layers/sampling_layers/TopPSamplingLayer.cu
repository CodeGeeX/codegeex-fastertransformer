#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <float.h>

#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/kernels/sampling_topk_kernels.h"
#include "src/fastertransformer/kernels/sampling_topp_kernels.h"
#include "src/fastertransformer/layers/sampling_layers/TopPSamplingLayer.h"
#include "src/fastertransformer/utils/logger.h"

namespace fastertransformer {

template<typename T>
void TopPSamplingLayer<T>::allocateBuffer()
{
    FT_CHECK(false);
}

template<typename T>
void TopPSamplingLayer<T>::allocateBuffer(size_t batch_size, size_t top_k, float top_p)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    invokeTopPSampling<T>(nullptr,  // workspace
                          sampling_workspace_size_,
                          cub_temp_storage_size_,
                          nullptr,  // output_ids
                          nullptr,  // sequence_length
                          nullptr,  // finished_buffer
                          nullptr,  // cum_log_probs
                          nullptr,  // output_log_probs
                          nullptr,  // log_probs
                          topp_id_vals_buf_,
                          topp_offset_buf_,
                          begin_topp_offset_buf_,
                          curandstate_buf_,
                          batch_size,
                          vocab_size_padded_,
                          nullptr,
                          top_p,
                          stream_,
                          cuda_device_prop_);

    sampling_workspace_ = allocator_->reMalloc(sampling_workspace_, sampling_workspace_size_, true);
    curandstate_buf_ = reinterpret_cast<hiprandState_t*>(
        allocator_->reMalloc(curandstate_buf_, sizeof(hiprandState_t) * batch_size, true));

    topp_id_vals_buf_ = reinterpret_cast<int*>(
        allocator_->reMalloc(topp_id_vals_buf_, sizeof(int) * batch_size * vocab_size_padded_, false));
    topp_offset_buf_ =
        reinterpret_cast<int*>(allocator_->reMalloc(topp_offset_buf_, sizeof(int) * (batch_size + 1), false));
    begin_topp_offset_buf_ =
        reinterpret_cast<int*>(allocator_->reMalloc(begin_topp_offset_buf_, sizeof(int) * (batch_size + 1), false));
    is_allocate_buffer_ = true;
}

template<typename T>
void TopPSamplingLayer<T>::freeBuffer()
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    if (is_allocate_buffer_ == true) {
        allocator_->free(sampling_workspace_);
        allocator_->free(curandstate_buf_);
        allocator_->free(topp_id_vals_buf_);
        allocator_->free(topp_offset_buf_);
        allocator_->free(begin_topp_offset_buf_);
        is_allocate_buffer_ = false;
    }
}

template<typename T>
void TopPSamplingLayer<T>::invokeInitialize(size_t batch_size,
                                            unsigned long long random_seed,
                                            hiprandState_t* curandstate_buf)
{
    FT_LOG_DEBUG(__PRETTY_FUNCTION__);
    invokeCurandInitialize(curandstate_buf, batch_size, random_seed, stream_);
    sync_check_cuda_error();
}

template<typename T>
void TopPSamplingLayer<T>::runSampling(std::vector<fastertransformer::Tensor>* output_tensors,
                                       const std::vector<fastertransformer::Tensor>* input_tensors)
{
    // input_tensors:
    //      logits [local_batch_size, vocab_size_padded]
    //      embedding_bias [vocab_size_padded]
    //      step [1] on cpu
    //      max_input_length [1] on cpu
    //      input_lengths [local_batch_size]
    //      ite [1] on cpu
    //      random_seed [1] on cpu

    // output_tensors:
    //      output_ids [max_seq_len, batch_size]
    //      finished [local_batch_size]
    //      sequence_length [local_batch_size]
    //      cum_log_probs [batch_size], must be float*
    //          The cumultative log probability of generated tokens.
    //      output_log_probs [local_batch_size], must be float*
    //          The log probs at the current step.

    FT_CHECK(false);  // TODO deprecated, need to remove
    std::unordered_map<std::string, Tensor> input_tensors_map{{"logits", input_tensors->at(0)},
                                                              {"embedding_bias", input_tensors->at(1)},
                                                              {"step", input_tensors->at(2)},
                                                              {"max_input_length", input_tensors->at(3)},
                                                              {"input_lengths", input_tensors->at(4)},
                                                              {"ite", input_tensors->at(5)}};
    if (input_tensors->size() == 7) {
        input_tensors_map.insert({"random_seed", input_tensors->at(6)});
    }

    std::unordered_map<std::string, Tensor> output_tensors_map{{"output_ids", output_tensors->at(0)},
                                                               {"finished", output_tensors->at(1)},
                                                               {"sequence_length", output_tensors->at(2)},
                                                               {"cum_log_probs", output_tensors->at(3)},
                                                               {"output_log_probs", output_tensors->at(4)}};
    runSampling(&output_tensors_map, &input_tensors_map);
}

template<typename T>
void TopPSamplingLayer<T>::runSampling(std::unordered_map<std::string, Tensor>* output_tensors,
                                       const std::unordered_map<std::string, Tensor>* input_tensors)
{
    // input_tensors:
    //      logits [local_batch_size, vocab_size_padded]
    //      embedding_bias [vocab_size_padded]
    //      step [1] on cpu
    //      max_input_length [1] on cpu
    //      input_lengths [local_batch_size]
    //      ite [1] on cpu
    //      runtime_top_p [1] or [batch_size] on cpu, optional
    //      temperature [1] or [batch_size] on cpu, optional
    //      len_penalty [1] or [batch_size] on cpu, optional
    //      repetition_penalty [1] or [batch_size] on cpu, optional
    //      random_seed [1] or [batch_size] on cpu, optional

    // output_tensors:
    //      output_ids [max_seq_len, batch_size]
    //      finished [local_batch_size]
    //      sequence_length [local_batch_size]
    //      cum_log_probs [batch_size], must be float*, optional
    //          The cumultative log probability of generated tokens.
    //      output_log_probs [local_batch_size], must be float*, optional
    //          The log probs at the current step.

    FT_CHECK(input_tensors->size() >= 6);
    FT_CHECK(output_tensors->size() >= 3);

    const int batch_size = output_tensors->at("output_ids").shape[1];
    const int local_batch_size = input_tensors->at("logits").shape[0];
    const int step = *((int*)input_tensors->at("step").data);
    const int ite = *((int*)input_tensors->at("ite").data);

    const float runtime_top_p = input_tensors->at("runtime_top_p").shape[0] == 1 ?
                                    input_tensors->at("runtime_top_p").getVal<float>(0) :
                                    input_tensors->at("runtime_top_p").getVal<float>(ite * local_batch_size);
    allocateBuffer(batch_size, 0, runtime_top_p);

    invokeTopPInitialize(
        topp_id_vals_buf_, topp_offset_buf_, begin_topp_offset_buf_, local_batch_size, vocab_size_padded_, stream_);
    sync_check_cuda_error();

    if (input_tensors->find("random_seed") != input_tensors->end()) {
        unsigned long long int random_seed =
            input_tensors->at("random_seed").shape[0] == 1 ?
                (unsigned long long int)input_tensors->at("random_seed").getVal<int>(0) :
                (unsigned long long int)input_tensors->at("random_seed").getVal<int>(ite * local_batch_size);
        invokeInitialize(local_batch_size, random_seed, curandstate_buf_ + ite * local_batch_size);
    }

    invokeAddBiasSoftMax((T*)(input_tensors->at("logits").data),
                         (T*)(nullptr),
                         (const int*)input_tensors->at("end_id").data,
                         (bool*)output_tensors->at("finished").data,
                         local_batch_size,
                         vocab_size_padded_,
                         vocab_size_,
                         stream_);
    sync_check_cuda_error();

    float* cum_log_probs =
        output_tensors->count("cum_log_probs") ? output_tensors->at("cum_log_probs").getPtr<float>() : nullptr;
    float* output_log_probs =
        output_tensors->count("output_log_probs") ? output_tensors->at("output_log_probs").getPtr<float>() : nullptr;
    invokeTopPSampling<T>(sampling_workspace_,
                          sampling_workspace_size_,
                          cub_temp_storage_size_,
                          ((int*)output_tensors->at("output_ids").data) + step * batch_size + ite * local_batch_size,
                          (int*)output_tensors->at("sequence_length").data,
                          (bool*)output_tensors->at("finished").data,
                          cum_log_probs,
                          output_log_probs,
                          (T*)(input_tensors->at("logits").data),
                          topp_id_vals_buf_,
                          topp_offset_buf_,
                          begin_topp_offset_buf_,
                          curandstate_buf_ + ite * local_batch_size,
                          local_batch_size,
                          vocab_size_padded_,
                          (const int*)input_tensors->at("end_id").data,
                          runtime_top_p,
                          stream_,
                          cuda_device_prop_);
    sync_check_cuda_error();
}

template<typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(size_t max_batch_size,
                                        size_t vocab_size,
                                        size_t vocab_size_padded,
                                        int end_id,
                                        float top_p,
                                        unsigned long long random_seed,
                                        float temperature,
                                        float len_penalty,
                                        float repetition_penalty,
                                        hipStream_t stream,
                                        cublasMMWrapper* cublas_wrapper,
                                        IAllocator* allocator,
                                        bool is_free_buffer_after_forward,
                                        hipDeviceProp_t* cuda_device_prop):
    BaseSamplingLayer<T>(max_batch_size,
                         vocab_size,
                         vocab_size_padded,
                         end_id,
                         0,
                         top_p,
                         random_seed,
                         temperature,
                         len_penalty,
                         repetition_penalty,
                         stream,
                         cublas_wrapper,
                         allocator,
                         is_free_buffer_after_forward,
                         cuda_device_prop)
{
}

template<typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(TopPSamplingLayer<T> const& top_p_sampling_layer):
    BaseSamplingLayer<T>(top_p_sampling_layer)
{
}

template<typename T>
TopPSamplingLayer<T>::~TopPSamplingLayer()
{
    freeBuffer();
}

template class TopPSamplingLayer<float>;
template class TopPSamplingLayer<half>;

}  // namespace fastertransformer
